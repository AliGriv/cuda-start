#include "hip/hip_runtime.h"
#include <iostream>
#include "funcs_cuda.cuh"
// Number of threads in each thread block
const int blockSize = 128;

void initializeCUDA(int argc, char **argv, int &devID)
{
    findCudaDevice(argc, (const char **)argv);
//    // By default, we use device 0, otherwise we override the device ID based on what is provided at the command line
//    hipError_t error;
//    devID = 0;
//
//    if (checkCmdLineFlag(argc, (const char **)argv, "device"))
//    {
//    devID = getCmdLineArgumentInt(argc, (const char **)argv, "device");
//    error = hipSetDevice(devID);
//
//    if (error != hipSuccess)
//    {
//    printf("hipSetDevice returned error code %d, line(%d)\n", error, __LINE__);
//    exit(EXIT_FAILURE);
//    }
//    }
//
//    // get number of SMs on this GPU
//    error = hipGetDevice(&devID);
//
//    if (error != hipSuccess)
//    {
//    printf("hipGetDevice returned error code %d, line(%d)\n", error, __LINE__);
//    exit(EXIT_FAILURE);
//    }
//
//    hipDeviceProp_t deviceProp;
//
//    error = hipGetDeviceProperties(&deviceProp, devID);
//
//    if (error != hipSuccess)
//    {
//    printf("hipGetDeviceProperties returned error code %d, line(%d)\n", error, __LINE__);
//    exit(EXIT_FAILURE);
//    }
//
//    printf("GPU Device %d: \"%s\" with compute capability %d.%d\n\n", devID, deviceProp.name, deviceProp.major, deviceProp.minor);

}
__global__ void VectorAdd_Kernel(const double *a, const double *b, double *c, const int n) {
    int id = blockIdx.x*blockDim.x+threadIdx.x;
    // Make sure we do not go out of bounds
    if (id < n) {
        c[id] = a[id] + b[id];
    }

}

void VectorAdd_GPU(const double *h_a, const double *h_b, double *h_c, const int n) {

    // Size, in bytes, of each vector
    size_t bytes = n*sizeof(double);
    // Device input vectors
    double *dd_a;
    double *dd_b;
    //Device output vector
    double *dd_c;
    // Allocate memory for each vector on GPU
    hipMalloc(&dd_a, bytes);
    hipMalloc(&dd_b, bytes);
    hipMalloc(&dd_c, bytes);
    // Copy host vectors to device
    hipMemcpy( dd_a, h_a, bytes, hipMemcpyHostToDevice);
    hipMemcpy( dd_b, h_b, bytes, hipMemcpyHostToDevice);

    // Number of threads in each thread block
//    int blockSize = 10000;

    // Number of thread blocks in grid
    int gridSize = (int)ceil((float)n/blockSize);

    // Execute the kernel
    VectorAdd_Kernel<<<gridSize, blockSize>>>(dd_a, dd_b, dd_c, n);
    hipMemcpy( h_c, dd_c, bytes, hipMemcpyDeviceToHost );
    // Release device memory
    hipFree(dd_a);
    hipFree(dd_b);
    hipFree(dd_c);
}

void VectorsClass::VectorAdd_GPU_InClass(double *h_a, double *h_b, double *h_c, int n) {
    // Copy host vectors to device
    hipMemcpy( this->d_a, h_a, bytes, hipMemcpyHostToDevice);
    hipMemcpy( this->d_b, h_b, bytes, hipMemcpyHostToDevice);

    // Number of threads in each thread block
//    int blockSize = 10000;

    // Number of thread blocks in grid
    int gridSize = (int)ceil((float)n/blockSize);

    // Execute the kernel
    VectorAdd_Kernel<<<gridSize, blockSize>>>(this->d_a, this->d_b, this->d_c, n);
    hipDeviceSynchronize();
    hipMemcpy( h_c, this->d_c, bytes, hipMemcpyDeviceToHost );
    // Release device memory

}
void VectorsClass::VectorAdd_GPU_InClass(double *h_c, int n) {

    // Number of thread blocks in grid
    int gridSize = (int)ceil((float)n/blockSize);

    // Execute the kernel
    VectorAdd_Kernel<<<gridSize, blockSize>>>(this->d_a, this->d_b, this->d_c, n);
    std::cout << "bytes is " << this->bytes << std::endl;
    checkCudaErrors(hipMemcpy( h_c, this->d_c, this->bytes, hipMemcpyDeviceToHost ));
//    for (int i {0}; i < 10; ++i) {
//        std::cout << h_c[i] << std::endl;
//    }
    // Release device memory
}